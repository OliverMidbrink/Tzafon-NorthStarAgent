#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        return -1;
    }
    
    printf("Found %d CUDA device(s):\n", deviceCount);
    
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device %d: %s\n", i, prop.name);
        printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
        printf("  Global Memory: %lu MB\n", prop.totalGlobalMem / (1024 * 1024));
        printf("  Multiprocessors: %d\n", prop.multiProcessorCount);
    }
    
    return 0;
}
